#include "hip/hip_runtime.h"
#include <type_traits>

#include <ATen/ATen.h>
#include <ATen/Dispatch.h>

#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/detail/KernelUtils.h>
#include <ATen/cuda/detail/IndexUtils.cuh>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/cuda/MemoryAccess.cuh>
#include <ATen/native/cuda/PersistentSoftmax.cuh>
#include <ATen/native/cuda/block_reduce.cuh>

#include <c10/cuda/CUDAMathCompat.h>
#include <c10/cuda/CUDAStream.h>

#include <ATen/native/nested/NestedTensorTransformerFunctions.h>

#include <c10/util/Half.h>

#define BLOCK_DIM 256
#define GRID_DIM_Y 16

namespace at {
namespace native {

template <typename T>
__global__ void remove_padding_transform0213_2(
    const T* input,
    T* output,
    const int* offsets,
    const int* input_sizes,
    const int* output_sizes,
    int output_dim,
    const int batch_size) {
  const int batch_id = blockIdx.x;
  const int grid_id = blockIdx.y;
  const int tid = threadIdx.x + grid_id * BLOCK_DIM;
  const int grainsize = GRID_DIM_Y * BLOCK_DIM;
  const int offset = offsets[batch_id];
  const int* sizes_i = output_sizes + batch_id * output_dim;
  const int numel_i = sizes_i[0] * sizes_i[1];
  int input_offset =
      batch_id * input_sizes[1] * input_sizes[2] * input_sizes[3];
  for (int ii = 0; ii < (numel_i / grainsize); ii++) {
    const int i = ii * grainsize + tid;
    const int i2 = i / sizes_i[1];
    const int i13 = i % sizes_i[1];
    const int i1 = i13 / (sizes_i[1] / input_sizes[1]);
    const int i3 = i13 % (sizes_i[1] / input_sizes[1]);

    output[offset + i] = input
        [input_offset + i1 * input_sizes[2] * input_sizes[3] +
         i2 * input_sizes[3] + i3];
  }
  const int i = (numel_i / grainsize) * grainsize + tid;
  if (i < numel_i) {
    const int i2 = i / sizes_i[1];
    const int i13 = i % sizes_i[1];
    const int i1 = i13 / (sizes_i[1] / input_sizes[1]);
    const int i3 = i13 % (sizes_i[1] / input_sizes[1]);
    output[offset + i] = input
        [input_offset + i1 * input_sizes[2] * input_sizes[3] +
         i2 * input_sizes[3] + i3];
  }
}

template <typename T>
__global__ void remove_padding_2(
    const T* input,
    T* output,
    const int* offsets,
    const int* input_sizes,
    const int* output_sizes,
    int output_dim,
    const int batch_size) {
  const int batch_id = blockIdx.x;
  const int grid_id = blockIdx.y;
  const int tid = threadIdx.x + grid_id * BLOCK_DIM;
  const int grainsize = GRID_DIM_Y * BLOCK_DIM;
  const int offset = offsets[batch_id];
  const int* sizes_i = output_sizes + batch_id * output_dim;
  const int numel_i = sizes_i[0] * sizes_i[1];
  int input_offset = batch_id * input_sizes[1] * input_sizes[2];
  for (int ii = 0; ii < (numel_i / grainsize); ii++) {
    const int i = ii * grainsize + tid;
    const int i0 = i / sizes_i[1];
    const int i1 = i % sizes_i[1];
    const int i0_offset = i0 * input_sizes[2];
    output[offset + i] = input[input_offset + i0_offset + i1];
  }
  const int i = (numel_i / grainsize) * grainsize + tid;
  if (i < numel_i) {
    const int i0 = i / sizes_i[1];
    const int i1 = i % sizes_i[1];
    const int i0_offset = i0 * input_sizes[2];
    output[offset + i] = input[input_offset + i0_offset + i1];
  }
}

template <typename T>
__global__ void remove_padding(
    const T* input,
    T* output,
    const int* offsets,
    const int* input_sizes,
    const int* output_sizes,
    int output_dim,
    const int batch_size) {
  const int batch_id = blockIdx.x;
  const int grid_id = blockIdx.y;
  const int tid = threadIdx.x + grid_id * BLOCK_DIM;
  const int grainsize = GRID_DIM_Y * BLOCK_DIM;
  const int offset = offsets[batch_id];
  const int* sizes_i = output_sizes + batch_id * output_dim;
  const int numel_i = sizes_i[0] * sizes_i[1] * sizes_i[2];
  int input_offset =
      batch_id * input_sizes[1] * input_sizes[2] * input_sizes[3];
  for (int ii = 0; ii < (numel_i / grainsize); ii++) {
    const int i = ii * grainsize + tid;
    const int i0 = i / (sizes_i[1] * sizes_i[2]);
    const int i1 = (i % (sizes_i[1] * sizes_i[2])) / sizes_i[2];
    const int i2 = i % sizes_i[2];
    const int i0_offset = i0 * input_sizes[2] * input_sizes[3];
    const int i1_offset = i1 * input_sizes[3];
    output[offset + i] = input[input_offset + i0_offset + i1_offset + i2];
  }
  const int i = (numel_i / grainsize) * grainsize + tid;
  if (i < numel_i) {
    const int i0 = i / (sizes_i[1] * sizes_i[2]);
    const int i1 = (i % (sizes_i[1] * sizes_i[2])) / sizes_i[2];
    const int i2 = i % sizes_i[2];
    const int i0_offset = i0 * input_sizes[2] * input_sizes[3];
    const int i1_offset = i1 * input_sizes[3];
    output[offset + i] = input[input_offset + i0_offset + i1_offset + i2];
  }
}

template <typename T>
void remove_padding_kernelLauncher(
    const T* input,
    T* output,
    const int* offsets,
    const int* input_sizes,
    const int* output_sizes,
    int output_dim,
    const int batch_size) {
  dim3 grid;
  grid.x = batch_size;
  grid.y = GRID_DIM_Y;
  at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();
  if (output_dim == 2) {
    remove_padding_2<T><<<grid, BLOCK_DIM, 0, stream>>>(
        input,
        output,
        offsets,
        input_sizes,
        output_sizes,
        output_dim,
        batch_size);
  } else {
    remove_padding<T><<<grid, BLOCK_DIM, 0, stream>>>(
        input,
        output,
        offsets,
        input_sizes,
        output_sizes,
        output_dim,
        batch_size);
  }
}

template <typename T>
void remove_padding_transform0213_kernelLauncher(
    const T* input,
    T* output,
    const int* offsets,
    const int* input_sizes,
    const int* output_sizes,
    int output_dim,
    const int batch_size) {
  dim3 grid;
  grid.x = batch_size;
  grid.y = GRID_DIM_Y;
  at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();
  TORCH_CHECK(
      output_dim == 2,
      "remove padding transform0213 only support output dim == 2");

  remove_padding_transform0213_2<T><<<grid, BLOCK_DIM, 0, stream>>>(
      input,
      output,
      offsets,
      input_sizes,
      output_sizes,
      output_dim,
      batch_size);
}

template void remove_padding_kernelLauncher<float>(
    const float* input,
    float* output,
    const int* offsets,
    const int* input_sizes,
    const int* output_sizes,
    int output_dim,
    const int batch_size);

template void remove_padding_kernelLauncher<c10::Half>(
    const c10::Half* input,
    c10::Half* output,
    const int* offsets,
    const int* input_sizes,
    const int* output_sizes,
    int output_dim,
    const int batch_size);

template void remove_padding_transform0213_kernelLauncher<float>(
    const float* input,
    float* output,
    const int* offsets,
    const int* input_sizes,
    const int* output_sizes,
    int output_dim,
    const int batch_size);

template void remove_padding_transform0213_kernelLauncher<c10::Half>(
    const c10::Half* input,
    c10::Half* output,
    const int* offsets,
    const int* input_sizes,
    const int* output_sizes,
    int output_dim,
    const int batch_size);

template <typename T>
__global__ void add_padding_1(
    const T* input,
    T* output,
    T padding_value,
    const int* offsets,
    const int* input_sizes,
    int input_dim,
    int output_sizes_1,
    const int batch_size) {
  const int batch_id = blockIdx.x;
  const int grid_id = blockIdx.y;
  const int tid = threadIdx.x + grid_id * BLOCK_DIM;
  const int grainsize = GRID_DIM_Y * BLOCK_DIM;
  const int* sizes_i = input_sizes + batch_id * input_dim;
  const int batch_output_offset = batch_id * output_sizes_1;
  for (int ii = 0; ii < (output_sizes_1 / grainsize); ii++) {
    const int i = ii * grainsize + tid;
    const int output_offset = batch_output_offset + i;
    if (batch_id < batch_size && i < sizes_i[0]) {
      const int batch_input_offset = offsets[batch_id];
      output[output_offset] = input[batch_input_offset + i];
    } else {
      output[output_offset] = padding_value;
    }
  }
  const int i = (output_sizes_1 / grainsize) * grainsize + tid;
  if (i < output_sizes_1) {
    const int output_offset = batch_output_offset + i;
    if (batch_id < batch_size && (i < sizes_i[0])) {
      const int batch_input_offset = offsets[batch_id];
      output[output_offset] = input[batch_input_offset + i];
    } else {
      output[output_offset] = padding_value;
    }
  }
}

template <typename T>
__global__ void add_padding_2(
    const T* input,
    T* output,
    T padding_value,
    const int* offsets,
    const int* input_sizes,
    int input_dim,
    int output_sizes_1,
    int output_sizes_2,
    const int batch_size) {
  const int batch_id = blockIdx.x;
  const int grid_id = blockIdx.y;
  const int tid = threadIdx.x + grid_id * BLOCK_DIM;
  const int grainsize = GRID_DIM_Y * BLOCK_DIM;
  const int* sizes_i = input_sizes + batch_id * input_dim;
  const int output_offset = batch_id * output_sizes_1 * output_sizes_2;
  const int output_numel = output_sizes_1 * output_sizes_2;
  for (int ii = 0; ii < (output_numel / grainsize); ii++) {
    const int i = ii * grainsize + tid;
    const int i0 = i / (output_sizes_2);
    const int i1 = i - i0 * output_sizes_2;
    if (batch_id < batch_size && i0 < sizes_i[0] && i1 < sizes_i[1]) {
      const int offset = offsets[batch_id];
      const int input_offset = offset + i0 * sizes_i[1] + i1;
      output[output_offset + i] = input[input_offset];
    } else {
      output[output_offset + i] = padding_value;
    }
  }
  const int i = (output_numel / grainsize) * grainsize + tid;
  if (i < output_numel) {
    const int i0 = i / (output_sizes_2);
    const int i1 = i - i0 * output_sizes_2;
    if (batch_id < batch_size && i0 < sizes_i[0] && i1 < sizes_i[1]) {
      const int offset = offsets[batch_id];
      const int input_offset = offset + i0 * sizes_i[1] + i1;
      output[output_offset + i] = input[input_offset];
    } else {
      output[output_offset + i] = padding_value;
    }
  }
}

template <typename T>
__global__ void add_padding_3(
    const T* input,
    T* output,
    T padding_value,
    const int* offsets,
    const int* input_sizes,
    int input_dim,
    int output_sizes_1,
    int output_sizes_2,
    int output_sizes_3,
    const int batch_size) {
  const int batch_id = blockIdx.x;
  const int grid_id = blockIdx.y;
  const int tid = threadIdx.x + grid_id * BLOCK_DIM;
  const int grainsize = GRID_DIM_Y * BLOCK_DIM;
  const int* sizes_i = input_sizes + batch_id * input_dim;
  const int output_offset =
      batch_id * output_sizes_1 * output_sizes_2 * output_sizes_3;
  const int output_numel = output_sizes_1 * output_sizes_2 * output_sizes_3;
  for (int ii = 0; ii < (output_numel / grainsize); ii++) {
    const int i = ii * grainsize + tid;
    const int i0 = i / (output_sizes_2 * output_sizes_3);
    const int i1 = (i % (output_sizes_2 * output_sizes_3)) / output_sizes_3;
    const int i2 = i % output_sizes_3;
    if (batch_id < batch_size && i0 < sizes_i[0] && i1 < sizes_i[1] && i2 < sizes_i[2]) {
      const int offset = offsets[batch_id];
      const int input_offset =
          offset + i0 * (sizes_i[1] * sizes_i[2]) + i1 * sizes_i[2] + i2;
      output[output_offset + i] = input[input_offset];
    } else {
      output[output_offset + i] = padding_value;
    }
  }
  const int i = (output_numel / grainsize) * grainsize + tid;
  if (i < output_numel) {
    const int i0 = i / (output_sizes_2 * output_sizes_3);
    const int i1 = (i % (output_sizes_2 * output_sizes_3)) / output_sizes_3;
    const int i2 = i % output_sizes_3;
    if (batch_id < batch_size && i0 < sizes_i[0] && i1 < sizes_i[1] && i2 < sizes_i[2]) {
      const int offset = offsets[batch_id];
      const int input_offset =
          offset + i0 * (sizes_i[1] * sizes_i[2]) + i1 * sizes_i[2] + i2;
      output[output_offset + i] = input[input_offset];
    } else {
      output[output_offset + i] = padding_value;
    }
  }
}

template <typename T>
void add_padding_kernelLauncher(
    T* input, // [batch_size x None]
    T* output, // [batch_size x max(input.nested_size(1)) x inner_size]
    T padding_value,
    const int* offsets,
    const int* input_sizes,
    int input_dim,
    const std::vector<int64_t>& output_sizes,
    const int batch_size,
    const int output_batch_size) {
  at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();
  dim3 grid;
  grid.x = output_batch_size;
  grid.y = GRID_DIM_Y;
  if (input_dim == 1) {
    add_padding_1<T><<<grid, BLOCK_DIM, 0, stream>>>(
        input,
        output,
        padding_value,
        offsets,
        input_sizes,
        input_dim,
        output_sizes[1],
        batch_size);
  }
  if (input_dim == 2) {
    add_padding_2<T><<<grid, BLOCK_DIM, 0, stream>>>(
        input,
        output,
        padding_value,
        offsets,
        input_sizes,
        input_dim,
        output_sizes[1],
        output_sizes[2],
        batch_size);
  }
  if (input_dim == 3) {
    add_padding_3<T><<<grid, BLOCK_DIM, 0, stream>>>(
        input,
        output,
        padding_value,
        offsets,
        input_sizes,
        input_dim,
        output_sizes[1],
        output_sizes[2],
        output_sizes[3],
        batch_size);
  }
}

template void add_padding_kernelLauncher<double>(
    double* input,
    double* output,
    double padding_value,
    const int* offsets,
    const int* input_sizes,
    int input_dim,
    const std::vector<int64_t>& output_sizes,
    const int batch_size,
    const int output_batch_size);

template void add_padding_kernelLauncher<float>(
    float* input,
    float* output,
    float padding_value,
    const int* offsets,
    const int* input_sizes,
    int input_dim,
    const std::vector<int64_t>& output_sizes,
    const int batch_size,
    const int output_batch_size);

template void add_padding_kernelLauncher<c10::Half>(
    c10::Half* input,
    c10::Half* output,
    c10::Half padding_value,
    const int* offsets,
    const int* input_sizes,
    int input_dim,
    const std::vector<int64_t>& output_sizes,
    const int batch_size,
    const int output_batch_size);

/// ************************** softmax for attention **************************
// softmax kernel code is copied from
// https://raw.githubusercontent.com/NVIDIA/FasterTransformer/main/fastertransformer/cuda/attention_kernels.cu

namespace {

#define FINAL_MASK 0xffffffff

template <typename T>
__inline__ __device__
T warpReduceSum(T val)
{
  for(int mask = 16; mask > 0; mask >>= 1)
    val += __shfl_xor_sync(FINAL_MASK, val, mask, 32);
  return val;
}

/* Calculate the sum of all elements in a block */
template <typename T>
  __inline__ __device__
T blockReduceSum(T val)
{
  static __shared__ T shared[32]; 
  int lane = threadIdx.x & 0x1f; 
  int wid = threadIdx.x >> 5;  

  val = warpReduceSum<T>(val);

  if(lane == 0)
    shared[wid] = val;

  __syncthreads();

  val = (threadIdx.x < (blockDim.x >> 5 )) ? shared[lane] : (T)(0.0f);
  val = warpReduceSum<T>(val);
                              
  return val;
}

template <typename T>
  __inline__ __device__
T warpReduceMax(T val)
{
  for(int mask = 16; mask > 0; mask >>= 1) {
    T tmp = __shfl_xor_sync(FINAL_MASK, val, mask, 32);
    if (tmp > val) {
      val = tmp;
    }
  }
  return val;
}

/* Calculate the maximum of all elements in a block */
template <typename T>
  __inline__ __device__
T blockReduceMax(T val)
{
  static __shared__ T shared[32]; 
  int lane = threadIdx.x & 0x1f; // in-warp idx
  int wid = threadIdx.x >> 5;  // warp idx

  val = warpReduceMax(val); // get maxx in each warp

  if(lane == 0) // record in-warp maxx by warp Idx
    shared[wid] = val;

  __syncthreads();


  val = (threadIdx.x < (blockDim.x >> 5 )) ? shared[lane] : 0;
  val = warpReduceMax(val);

  return val;
}

template <typename T>
__global__
void softmax_kernel(T* qk_buf_, const T* attr_mask, const int batch_size, const int head_num, const int seq_len, 
  const T scalar)
{
    int batch_id = blockIdx.x / head_num;
    int qk_offset = blockIdx.x * seq_len * seq_len;
    int mask_offset = batch_id * seq_len * seq_len;

    __shared__ float s_sum, s_max;

    for(int i = 0; i < seq_len; ++i)
    {
      float qk = threadIdx.x < seq_len ? (float)qk_buf_[threadIdx.x + qk_offset] : 0.0f;
      float mask_val = threadIdx.x < seq_len ? (float)attr_mask[threadIdx.x + mask_offset] : 0.0f;
      
      mask_val = (1.0f - mask_val) * -10000.0f;

      float tmp = threadIdx.x < seq_len ? (float)(qk * (float)scalar + mask_val): -1e20f;

      float max_val = blockReduceMax<float>(tmp);

      if(threadIdx.x == 0)
        s_max = max_val;
      __syncthreads();

      qk = threadIdx.x < seq_len ? __expf(tmp - s_max) : 0.0f;

      float sum_val = blockReduceSum<float>(qk);

      if(threadIdx.x == 0)
      {
        s_sum = sum_val + 1e-6f;
      }
      __syncthreads();

      if(threadIdx.x < seq_len)
        qk_buf_[threadIdx.x + qk_offset] = (T)(qk / s_sum);

      qk_offset += seq_len;
      mask_offset += seq_len;
    }
}

}

// Changed this align with prior API
// Renamed and switched head_num with seq_len
template<typename T>
void softmax_kernel_kernelLauncher(
  T* buffer,
  const T* attr_mask,
  const int batch_size,
  const int head_num,
  const int seq_len,
  const T scalar,
  hipStream_t stream)
{
  dim3 grid, block;
  if(seq_len <= 32) {
    block.x = 32;
  } else if(seq_len > 32 && seq_len <= 64) {
    block.x = 64;
  } else if(seq_len > 64 && seq_len <= 128) {
    block.x = 128;
  } else if(seq_len > 128 && seq_len <= 256) {
    block.x = 256;
  } else if(seq_len > 256 && seq_len <= 512) {
    block.x = 512;
  } else {
    block.x = 1024;
  }

  grid.x = batch_size * head_num;
  softmax_kernel<T><<<grid, block, 0, stream>>>(buffer, attr_mask, batch_size, head_num, seq_len, scalar);
}

template void softmax_kernel_kernelLauncher<float>(
    float* qk_buf_, const float* attr_mask, 
    const int batch_size, const int head_num, const int seq_len, 
    const float scaler,
    const hipStream_t stream);

template void softmax_kernel_kernelLauncher<c10::Half>(
    c10::Half* qk_buf_, const c10::Half* attr_mask, 
    const int batch_size, const int head_num, const int seq_len, 
    const c10::Half scaler,
    const hipStream_t stream);

/// *********************************** fin ***********************************

} // namespace native
} // namespace at
